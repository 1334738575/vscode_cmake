
#include <hip/hip_runtime.h>
#include<stdio.h>

const int gsx = 1;
const int gsy = 1;
const int bsx = 2;
const int bsy = 4;
const int bsz = 1;
const int ts = gsx*gsy*bsx*bsy*bsz;

__global__ void hello_from_gpu(const int const* in1, const int const* in2, int* out){
	
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	const int yid = threadIdx.y;
	const int id = bid * bsx * bsy + tid * bsy + yid;
	out[id] = in1[id] * in2[id];
    printf("Thread %d: %d * %d = %d\n", id, in1[id], in2[id], out[id]);
	printf("hello world from gpu block %d and thread (%d,%d)!\n", bid, tid, yid);
}

extern "C" int mainCUDA()
{
    printf("helloword\n");
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount); //获取GPU数量
    printf("Size of GPU is %d", iDeviceCount);
    // int iDev = 0;
    // error = cudaSetDevice(iDev); //设置运行GPU编号
    
	size_t dataSize = ts * sizeof(int);
    int *a, *b, *s;
    a = (int *)malloc(dataSize);
    b = (int *)malloc(dataSize);
    s = (int *)malloc(dataSize);
    if(a != NULL && b != NULL && s != NULL){
        memset(a, 0, dataSize);
        memset(b, 0, dataSize);
        memset(s, 0, dataSize);
        printf("Host data prepear!\n");
    }

    int *ca, *cb, *cs;
    hipMalloc((int**)&ca, dataSize);
    hipMalloc((int**)&cb, dataSize);
    hipMalloc((int**)&cs, dataSize);
    if(ca != NULL && cb != NULL && cs != NULL){
        hipMemset(ca, 0, dataSize);
        hipMemset(cb, 0, dataSize);
        hipMemset(cs, 0, dataSize);
        printf("Device data prepear!\n");
    }

    a[0] = 1;
    a[1] = 2;
    a[2] = 3;
    a[3] = 4;
    a[4] = 4;
    a[5] = 3;
    a[6] = 2;
    a[7] = 1;
    b[0] = 9;
    b[1] = 8;
    b[2] = 7;
    b[3] = 6;
    b[4] = 6;
    b[5] = 7;
    b[6] = 8;
    b[7] = 9;
    // for(int i=0; i<ts; ++i){
	// 	printf("%d ", a[i]);
	// 	// printf("%d ", a[i]*b[i]);
	// }

    hipMemcpy(ca, a, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(cb, b, dataSize, hipMemcpyHostToDevice);
    // cudaMemcpy(s, cs, dataSize, cudaMemcpyHostToDevice);

	const dim3 grid_size(gsx, gsy, 1);
	const dim3 block_size(bsx, bsy, bsz);
	hello_from_gpu<<<grid_size,block_size>>>(ca,cb,cs);
	hipDeviceSynchronize();

    // cudaMemcpy(a, ca, dataSize, cudaMemcpyHostToDevice);
    // cudaMemcpy(b, cb, dataSize, cudaMemcpyHostToDevice);
    hipMemcpy(s, cs, dataSize, hipMemcpyDeviceToHost);

	for(int i=0; i<ts; ++i){
		printf("%d ", s[i]);
		// printf("%d ", a[i]*b[i]);
	}

    free(a);
    free(b);
    free(s);
    hipFree(ca);
    hipFree(cb);
    hipFree(cs);

    hipDeviceReset();
    return 0;
}
